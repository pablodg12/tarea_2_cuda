
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

void printMatrix(const int *A, int rows, int cols) {
    for(int i = 0; i < rows*cols*4; i++){
        printf("%d ", A[i]);   
        printf(" ");
        if ((i+1)%9 == 0){
          printf("|");
        }
    }
    printf("\n");
};

void readInput_soa(const char *filename, int **Soa,  int *rows, int *cols) {
  FILE *file;
  file = fopen(filename, "r");
  fscanf(file, "%d %d", rows, cols);
  int * A_F0 = (int *) malloc(*rows * (*cols)* (4) * sizeof(int));
  for(int i = 0; i < *rows*(*cols)*(4); i++) { 
    fscanf(file, "%d ", &A_F0[i]); 
  }
  fclose(file);
  *Soa = A_F0;
};

__global__ void step_periodic(int * array,int *buffer,int rows, int cols){
  int tId = threadIdx.x + blockIdx.x * blockDim.x;
if(tId < rows*cols){
    int x = tId%(cols);
    int y = (int) tId/rows;

    int c_aux = x -1;
    if (c_aux < 0){
      c_aux = cols -1;
    }
    if (buffer[(y*cols + c_aux) + rows*cols] == 1 && buffer[(y*cols + c_aux) + 3*rows*cols] == 1){
       array[tId] = 1;
    }else if (buffer[(y*cols + c_aux)] == 1 && buffer[(y*cols + c_aux) + 2*rows*cols] == 1){
      array[tId] = 0;
    }else if (buffer[(y*cols + c_aux)] == 1){
        array[tId] = 1;
    }else if (buffer[(y*cols + c_aux)] == 0){
      array[tId] = 0;
    }

    c_aux = x + 1;
    if (c_aux == cols){
      c_aux = 0;
    }

    if (buffer[(y*cols + c_aux) + rows*cols] == 1 && buffer[(y*cols + c_aux) + 3*rows*cols] == 1){
       array[tId+2*rows*cols] = 1;
    }else if (buffer[(y*cols + c_aux)] == 1 && buffer[(y*cols + c_aux) + 2*rows*cols] == 1){
      array[tId+2*rows*cols] = 0;
    }else if (buffer[(y*cols + c_aux)+ 2*rows*cols] == 1){
        array[tId+2*rows*cols] = 1;
    }else if (buffer[(y*cols + c_aux)+ 2*rows*cols] == 0){
      array[tId+2*rows*cols] = 0;
    }
    c_aux = y - 1;
    if (c_aux <0){
      c_aux = rows-1;
    }

    c_aux = (((y-1)%rows)+rows)%rows*cols;

    if (buffer[(c_aux + x)] == 1 && buffer[(c_aux + x) + 2*rows*cols] == 1){
       array[tId+rows*cols] = 1;
    }else if (buffer[(c_aux + x)+rows*cols] == 1 && buffer[(c_aux + x) + 3*rows*cols] == 1){
      array[tId+rows*cols] = 0;
    }else if (buffer[ (c_aux + x) + rows*cols ] == 1){
      array[tId+rows*cols] = 1;
    }else if (buffer[ (c_aux + x) + rows*cols ] == 0){
      array[tId+rows*cols] = 0;
    }

    c_aux = (((y+1)%rows)*cols);

    if (buffer[(c_aux + x)] == 1 && buffer[(c_aux + x) + 2*rows*cols] == 1){
       array[tId+3*rows*cols] = 1;
    }else if (buffer[(c_aux + x)+rows*cols] == 1 && buffer[(c_aux + x) + 3*rows*cols] == 1){
      array[tId+3*rows*cols] = 0;
    }else if (buffer[ (c_aux + x) + 3*rows*cols ] == 1){
      array[tId+3*rows*cols] = 1;
    }else if (buffer[ (c_aux + x) + 3*rows*cols ] == 0){
      array[tId+3*rows*cols] = 0;
    }
  }
 }  
int main(int argc, char const *argv[])
{
  int rows, cols;
  int *array;
  int *d_array;
  int *d_buffer;
  readInput_soa("../initial.txt", &array, &rows, &cols);
  int n = (int)(rows*cols);
  int block_size = 256;
  int grid_size = (int) ceil((float) n/ block_size);

  hipMalloc(&d_array ,4*rows * cols * sizeof(int));
  hipMalloc(&d_buffer,4*rows*cols*sizeof(int));
  hipMemcpy(d_array, array,4* rows * cols * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_buffer, array,4* rows * cols * sizeof(int), hipMemcpyHostToDevice);
  for(int k = 0; k < 1000; k++){
    step_periodic<<<grid_size, block_size>>>(d_array, d_buffer, rows, cols);
    hipMemcpy(d_buffer,d_array,4*rows*cols * sizeof(int), hipMemcpyDeviceToDevice);
  }
  hipMemcpy(array, d_array, 4*rows * cols * sizeof(int), hipMemcpyDeviceToHost);
  hipFree(d_array);
  hipFree(d_buffer);

  return(0);
}
