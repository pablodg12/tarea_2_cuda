
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

void printMatrix(const int *A, int rows, int cols) {
    for(int i = 0; i < rows*cols*4; i++){
        printf("%d ", A[i]);   
        printf(" ");
        if ((i+1)%9 == 0){
          printf("|");
        }
    }
    printf("\n");
};

void readInput_soa(const char *filename, int **Soa,  int *rows, int *cols) {
  FILE *file;
  file = fopen(filename, "r");
  fscanf(file, "%d %d", rows, cols);
  int * A_F0 = (int *) malloc(*rows * (*cols)* (4) * sizeof(int));
  for(int i = 0; i < *rows*(*cols)*(4); i++) { 
    fscanf(file, "%d ", &A_F0[i]); 
  }
  fclose(file);
  *Soa = A_F0;
};

__global__ void step_periodic_Soa(int * array,int rows, int cols){
  int tId = threadIdx.x + blockIdx.x * blockDim.x;
  if (tId < rows*cols){

    int x = tId%(cols);
    int y = (int) tId/rows;

    //Colission
    if (array[tId] == 1 && array[tId + 2*rows*cols] == 1){
      if(array[tId + rows*cols] == 0 && array[tId+rows*cols*3] == 0){
        array[tId] = 0;
        array[tId + 2*rows*cols] = 0;
        array[tId + rows*cols] = 1;
        array[tId+rows*cols*3] = 1;
      }
    }
    if (array[tId + rows*cols] == 1 && array[tId+rows*cols*3] == 1){
      if(array[tId] == 0 && array[tId + 2*rows*cols] == 0){
        array[tId + rows*cols] = 0;
        array[tId+rows*cols*3] = 0;
        array[tId] = 1;
        array[tId + 2*rows*cols] = 1;
      }
    }
    //streaming

    //right
    int c_aux = x + 1; 
    if (c_aux == cols){
        c_aux = 0;
      }
    if (array[tId] == 1){
        array[(y*rows + c_aux)] = array[tId]*2;
    }
    //left
    c_aux = x - 1;
    if (c_aux < 0){
      c_aux = cols -1;
    }
    if (array[tId+ 2*rows*cols] == 1){
      array[(y*rows + c_aux) + 2*rows*cols] = array[tId+ 2*rows*cols]*2;
    }

    //top
    c_aux = y + 1;
    if (c_aux == rows){
      c_aux = 0;
    }
    if (array[tId + rows*cols] == 1){
      array[(c_aux*rows + x) + + rows*cols] = array[tId+ rows*cols]*2;
    }

    //bottom
    c_aux = y + 1;
    if (c_aux < 0){
      c_aux = rows-1;
    }
    if (array[tId+ 3*rows*cols] == 1){
      array[(c_aux*rows + x)+ 3*rows*cols] = array[tId+ 3*rows*cols]*2;
    }

    //Correction
    if(array[tId] == 1){
      array[tId] = 0;
    }
    if(array[tId] == 2){
      array[tId] = 1;
    }
    if(array[tId+ rows*cols] == 1){
      array[tId+ rows*cols] = 0;
    }
    if(array[tId+ rows*cols] == 2){
      array[tId+ rows*cols] = 1;
    }
    if(array[tId+ 2*rows*cols] == 1){
      array[tId+ 2*rows*cols] = 0;
    }
    if(array[tId+ 2*rows*cols] == 2){
      array[tId+ 2*rows*cols] = 1;
    }
    if(array[tId+ 3*rows*cols] == 1){
      array[tId+ 3*rows*cols] = 0;
    }
    if(array[tId+ 3*rows*cols] == 2){
      array[tId+ 3*rows*cols] = 1;
    }

  }
};

int main(int argc, char const *argv[])
{
  int rows, cols;
  int *Soa;
  int *d_Soa;

  readInput_soa("../initial.txt", &Soa, &rows, &cols);

  //printMatrix(Soa,rows,cols);

  int n = (int)(rows*cols);
  int block_size = 256;
  int grid_size = (int) ceil((float)n / block_size);

  hipMalloc(&d_Soa, 4 * rows * cols * sizeof(int));
  hipMemcpy(d_Soa, Soa, 4 * rows * cols * sizeof(int), hipMemcpyHostToDevice);

  for(int k = 0; k < 1000; k++){
    step_periodic_Soa<<<grid_size, block_size>>>(d_Soa, rows, cols);
  }

  hipMemcpy(Soa, d_Soa, 4 * rows * cols * sizeof(int), hipMemcpyDeviceToHost);
  hipFree(d_Soa);


return 0;

}

