
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

void printMatrix(const int *A, int rows, int cols) {
    for(int i = 0; i < rows*cols*4; i++){
        printf("%d ", A[i]);   
        printf(" ");
        if ((i+1)%9 == 0){
          printf("|");
        }
    }
    printf("\n");
};

void readInput_soa(const char *filename, int **Soa,  int *rows, int *cols) {
  FILE *file;
  file = fopen(filename, "r");
  fscanf(file, "%d %d", rows, cols);
  int * A_F0 = (int *) malloc(*rows * (*cols)* (4) * sizeof(int));
  for(int i = 0; i < *rows*(*cols)*(4); i++) { 
    fscanf(file, "%d ", &A_F0[i]); 
  }
  fclose(file);
  *Soa = A_F0;
};

__global__ void step_periodic_Soa(int * array,int rows, int cols){
  extern __shared__ int buffer[];
  int tId = threadIdx.x + blockIdx.x * blockDim.x;
  if(threadIdx.x < 256){
    for(int i = threadIdx.x; i < rows*cols; i+=256 ){
      if (array[i] == 1 && array[i + 2*rows*cols] == 1){
        if(array[i + rows*cols] == 0 && array[i+rows*cols*3] == 0){
          buffer[i] = 0;
          buffer[i + 2*rows*cols] = 0;
          buffer[i + rows*cols] = 1;
          buffer[i+rows*cols*3] = 1;
        }
      }else if (array[i + rows*cols] == 1 && array[i+rows*cols*3] == 1){
        if(array[i] == 0 && array[i + 2*rows*cols] == 0){
          buffer[i + rows*cols] = 0;
          buffer[i+rows*cols*3] = 0;
          buffer[i] = 1;
          buffer[i + 2*rows*cols] = 1;
        }
      }else{
          buffer[i + rows*cols] = array[i + rows*cols];
          buffer[i+rows*cols*3] = array[i+rows*cols*3];
          buffer[i]  = array[i];
          buffer[i + 2*rows*cols] = array[i+2*rows*cols];
      }
    }
  }
  __syncthreads();

  //if(tId == 1){
   // for(int i = 0; i < rows*cols*4;i++){
   //   printf("%d ", buffer[i]);
   //   printf(" ");
   //   if ((i+1)%9 == 0){
  //        printf("|");
  //      }
 // }
//  printf("\n");
//}

if(tId < rows*cols){
    int x = tId%(cols);
    int y = (int) tId/rows;


    int c_aux = x -1;
    if (c_aux < 0){
      c_aux = cols -1;
    }
    if (buffer[(y*rows + c_aux)] == 1){
        array[tId] = 1;
    }else if (buffer[(y*rows + c_aux)] == 0){
      array[tId] = 0;
    }
    c_aux = x + 1;
    if (c_aux == cols){
      c_aux = 0;
    }
    if (buffer[(y*rows + c_aux) + 2*rows*cols ] == 1){
      array[tId+ 2*rows*cols] = 1;
    }else if (buffer[(y*rows + c_aux) + 2*rows*cols ] == 0){
      array[tId+ 2*rows*cols] = 0;
    }
    c_aux = y - 1;
    if (c_aux <0){
      c_aux = rows-1;
    }
    if (buffer[ (c_aux*rows + x) + rows*cols ] == 1){
      array[tId + rows*cols] = 1;
    }else if (buffer[ (c_aux*rows + x) + rows*cols ] == 0){
      array[tId + rows*cols] = 0;
    }
    c_aux = y + 1;
    if (c_aux == rows){
      c_aux = 0;
    }
    if (buffer[ (c_aux*rows + x)+ 3*rows*cols ] == 1){
      array[tId+ 3*rows*cols] = 1;
    }else if (buffer[ (c_aux*rows + x) + 3*rows*cols ] == 0){
      array[tId + 3*rows*cols] = 0;
    }
  }
}


int main(int argc, char const *argv[])
{
  int rows, cols;
  int *Soa;
  int *d_Soa;

  readInput_soa("../initial.txt", &Soa, &rows, &cols);

  //printMatrix(Soa,rows,cols);

  int n = (int)(rows*cols);
  int block_size = 256;
  int grid_size = (int) ceil((float)n / block_size);

  hipMalloc(&d_Soa, 4 * rows * cols * sizeof(int));
  hipMemcpy(d_Soa, Soa, 4 * rows * cols * sizeof(int), hipMemcpyHostToDevice);

  for(int k = 0; k < 1000; k++){
    step_periodic_Soa<<<grid_size, block_size,rows*cols*4>>>(d_Soa, rows, cols);
  }

  hipMemcpy(Soa, d_Soa, 4 * rows * cols * sizeof(int), hipMemcpyDeviceToHost);
  hipFree(d_Soa);


return 0;

}

