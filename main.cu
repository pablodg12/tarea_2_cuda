
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

void printMatrix(const int *A, int rows, int cols) {
    for(int i = 0; i < rows*cols*4; i++){
        printf("%d ", A[i]);   
        printf(" ");
        if ((i+1)%4 == 0){
          printf("|");
        }
    }
    printf("\n");
};

void readInput_soa(const char *filename, int **Soa,  int *rows, int *cols) {
  FILE *file;
  file = fopen(filename, "r");
  fscanf(file, "%d %d", rows, cols);
  int * A_F0 = (int *) malloc(*rows * (*cols)* (4) * sizeof(int));
  for(int i = 0; i < *rows*(*cols)*(4); i++) { 
    fscanf(file, "%d ", &A_F0[i]); 
  }
  fclose(file);
  *Soa = A_F0;
};

void readInput_aos(const char *filename, int **Aos, int *rows, int *cols) {
  FILE *file;
  file = fopen(filename, "r");
  fscanf(file, "%d %d", rows, cols);
  int * A_F1 = (int *) malloc(*rows * (*cols)* (4) * sizeof(int));
  for(int j = 0; j <  4; j++) {
    int counter = 0;
    for(int i = 0; i < *cols*(*rows); i++){
      fscanf(file, "%d ", &A_F1[counter +j]);
      counter = counter + 4;
      }
  }
  fclose(file);
  *Aos = A_F1;
}

//Periodic boundaries condition Array of Structures

__global__ void step_periodic_Aos(int * array,int rows, int cols){
  int tId = threadIdx.x + blockIdx.x * blockDim.x;
  if (tId < rows*cols){
    int x = tId%(cols);
    int y = (int) tId/rows;

    //Colission

    if (array[tId*4+0] == 1 && array[tId*4+2] == 1){
      if(array[tId*4+1] == 0 && array[tId*4+3] == 0){
        array[tId*4+0] = 0;
        array[tId*4+2] = 0;
        array[tId*4+1] = 1;
        array[tId*4+3] = 1;
      }
    }
    if (array[tId*4+1] == 1 && array[tId*4+3] == 1){
      if(array[tId*4+0] == 0 && array[tId*4+2] == 0){
        array[tId*4+1] = 0;
        array[tId*4+3] = 0;
        array[tId*4+0] = 1;
        array[tId*4+2] = 1;
      }
    }

    //streaming 

    //right
    int c_aux = x + 1;
    if (c_aux == cols){
      c_aux = 0;
    }
    if (array[tId*4] == 1){
      array[(y*rows + c_aux)*4] = array[tId*4]*2;
    }

    //left
    c_aux = x - 1;
    if (c_aux < 0){
      c_aux = cols -1;
    }
    if (array[tId*4+2] == 1){
      array[(y*rows + c_aux)*4 + 2] = array[tId*4+2]*2;
    }

    //top
    c_aux = y + 1;
    if (c_aux == rows){
      c_aux = 0;
    }
    if (array[tId*4+1] == 1){
      array[(c_aux*rows + x)*4 + 1] = array[tId*4+1]*2;
    }

    //bottom
    c_aux = y + 1;
    if (c_aux < 0){
      c_aux = rows-1;
    }
    if (array[tId*4+3] == 1){
      array[(c_aux*rows + x)*4 + 3] = array[tId*4+3]*2;
    }

    //Correction
    for(int i = 0; i < 4; i++){
      if(array[tId*4+i] == 1){
        array[tId*4+i] = 0;
      }
      if(array[tId*4+i] == 2){
        array[tId*4+i] = 1;
      }
    };
  }
};  

int main(int argc, char const *argv[])
{
  int rows, cols;
  int *Aos, *Soa;
  int *d_Aos, *d_Soa;

  readInput_aos("initial.txt", &Aos, &rows, &cols);
  readInput_soa("initial.txt", &Soa, &rows, &cols);

  //printMatrix(Aos,rows,cols);

  int n = (int)(rows*cols);
  int block_size = 256;
  int grid_size = (int) ceil((float)n / block_size);

  hipMalloc(&d_Aos, 4 * rows * cols * sizeof(int));
  hipMalloc(&d_Soa, 4 * rows * cols * sizeof(int));

  hipMemcpy(d_Aos, Aos, 4 * rows * cols * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_Soa, Soa, 4 * rows * cols * sizeof(int), hipMemcpyHostToDevice);

  for(int k = 0; k < 1000; k++){
    step_periodic_Aos<<<grid_size, block_size>>>(d_Aos, rows, cols);
  }

  hipMemcpy(Aos, d_Aos, 4 * rows * cols * sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(Soa, d_Soa, 4 * rows * cols * sizeof(int), hipMemcpyDeviceToHost);

  hipFree(d_Aos);
  hipFree(d_Soa);
  //printf("----- \n");
  //printMatrix(Aos,rows,cols);


  //printMatrix(Aos,rows,cols);
  //printf("----- \n");
  //printMatrix(Soa,rows,cols);

return 0;

}