
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

void readInput(const char *filename, int **Aos, int *rows, int *cols) {
  FILE *file;
  file = fopen(filename, "r");
  
  fscanf(file, "%d %d", rows, cols);
  int * A_F1 = (int *) malloc(*rows * (*cols)* (4) * sizeof(int));
  int * A_F2 = (int *) malloc(*rows * (*cols) * sizeof(int));
  for(int j = 0; j <  4; j++) {
    int counter = 0;
    for(int i = 0; i < *cols*(*rows); i++){
      fscanf(file, "%d ", &A_F1[counter +j]);
      counter = counter + 4;
      }
  }
  int counter = 0;
  for(int j = 0; j < *cols*(*rows);j++){
    A_F2[j] = A_F1[counter]*1 + A_F1[counter+1]*2 + A_F1[counter+2]*2*2 + A_F1[counter+3]*2*2*2;
    counter = counter +4;
  }
  *Aos = A_F2;
}

void printMatrix(int *A, int rows, int cols) {
    for(int i = 0; i < rows*cols; i++){
        printf("%i ", A[i]);
    }
    printf("\n");
};

__global__ void step_periodic(int * array,int rows, int cols){
  extern __shared__ int buffer[];
  int tId = threadIdx.x + blockIdx.x * blockDim.x;
  if(threadIdx.x < 256){
    for(int i = threadIdx.x; i < rows*cols; i+=256 ){
      if(array[i] == 10){
        buffer[i] = 5;
      }else if (array[i] == 5){
        buffer[i] = 10;
      }else{
        buffer[i] = array[i];
      }
    }
  }
   __syncthreads();
  //if(tId == 1){
   // for(int i = 0; i < rows*cols;i++){
   //   printf("%d ", buffer[i]);
   // }
  //  printf("\n");
  //}


  if (tId < rows*cols){
    int x = tId%(cols);
    int y = (int) tId/rows;
    int total = 0;

    int c_aux = x -1;
    if (c_aux < 0){
      c_aux = cols -1;
    }
    if (buffer[(y*rows + c_aux)] == 1 || buffer[(y*rows + c_aux)] == 3 || buffer[(y*rows + c_aux)] == 5 || 
        buffer[(y*rows + c_aux)] == 9 || buffer[(y*rows + c_aux)] == 7 || buffer[(y*rows + c_aux)] == 11 || 
        buffer[(y*rows + c_aux)] == 13 || buffer[(y*rows + c_aux)] == 15 ){
      total = total + 1;
    }else {
      total = total + 0;
    }
    c_aux = x + 1;
    if (c_aux == cols){
      c_aux = 0;
    }
    if (buffer[(y*rows + c_aux)] == 4 || buffer[(y*rows + c_aux)] == 5 || buffer[(y*rows + c_aux)] == 6 || 
        buffer[(y*rows + c_aux)] == 12 || buffer[(y*rows + c_aux)] == 7 || buffer[(y*rows + c_aux)] == 13 || 
        buffer[(y*rows + c_aux)] == 14 || buffer[(y*rows + c_aux)] == 15 ){
      total = total + 4;
    }else {
      total = total + 0;
    }
    c_aux = y - 1;
    if (c_aux <0){
      c_aux = rows-1;
    }
    if (buffer[(c_aux*rows + x)] == 2 || buffer[(c_aux*rows + x)] == 3 || buffer[(c_aux*rows + x)] == 6 || 
        buffer[(c_aux*rows + x)] == 10 || buffer[(c_aux*rows + x)] == 7 || buffer[(c_aux*rows + x)] == 11 || 
        buffer[(c_aux*rows + x)] == 14 || buffer[(c_aux*rows + x)] == 15 ){
      total = total + 2;
    }else {
      total = total + 0;
    }
    c_aux = y + 1;
    if (c_aux == rows){
      c_aux = 0;
    }
    if (buffer[(c_aux*rows + x)] == 8 || buffer[(c_aux*rows + x)] == 12 || buffer[(c_aux*rows + x)] == 10 || 
        buffer[(c_aux*rows + x)] == 9 || buffer[(c_aux*rows + x)] == 14 || buffer[(c_aux*rows + x)] == 13 || 
        buffer[(c_aux*rows + x)] == 11 || buffer[(c_aux*rows + x)] == 15 ){
      total = total + 8;
    }else {
      total = total + 0;
    }
    array[tId] = total;    
  }
}

int main(int argc, char const *argv[])
{
  int rows, cols;
  int *array;
  int *d_array;

  readInput("../initial.txt", &array, &rows, &cols);
  //printMatrix(array,rows,cols);

  int n = (int)(rows*cols);
  int block_size = 256;
  int grid_size = (int) ceil((float)n / block_size);

  hipMalloc(&d_array ,rows * cols * sizeof(int));
  hipMemcpy(d_array, array, rows * cols * sizeof(int), hipMemcpyHostToDevice);
  for(int k = 0; k < 1000; k++){
    step_periodic<<<grid_size, block_size,rows*cols>>>(d_array, rows, cols);
  }
  hipMemcpy(array, d_array, rows * cols * sizeof(int), hipMemcpyDeviceToHost);
  hipFree(d_array);

  return(0);
}
