
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

void printMatrix(const int *A, int rows, int cols) {
    for(int i = 0; i < rows*cols*4; i++){
        printf("%d ", A[i]);   
        printf(" ");
        if ((i+1)%4 == 0){
          printf("|");
        }
    }
    printf("\n");
};

void readInput_aos(const char *filename, int **Aos, int *rows, int *cols) {
  FILE *file;
  file = fopen(filename, "r");
  fscanf(file, "%d %d", rows, cols);
  int * A_F1 = (int *) malloc(*rows * (*cols)* (4) * sizeof(int));
  for(int j = 0; j <  4; j++) {
    int counter = 0;
    for(int i = 0; i < *cols*(*rows); i++){
      fscanf(file, "%d ", &A_F1[counter +j]);
      counter = counter + 4;
      }
  }
  fclose(file);
  *Aos = A_F1;
}

__global__ void step_periodic(int * array,int *buffer,int rows, int cols){
  int tId = threadIdx.x + blockIdx.x * blockDim.x;

if(tId < rows*cols){
    int x = tId%(cols);
    int y = (int) tId/rows;

    int c_aux = x -1;
    if (c_aux < 0){
      c_aux = cols -1;
    }
    if (buffer[(y*rows + c_aux)*4 + 1] == 1 && buffer[(y*rows + c_aux)*4 + 3] == 1){
       array[tId*4] = 1;
    }else if (buffer[(y*rows + c_aux)*4] == 1 && buffer[(y*rows + c_aux) + 2] == 1){
      array[tId*4] = 0;
    }else if (buffer[(y*rows + c_aux)*4] == 1){
        array[tId*4] = 1;
    }else if (buffer[(y*rows + c_aux)*4] == 0){
      array[tId*4] = 0;
    }

    c_aux = x + 1;
    if (c_aux == cols){
      c_aux = 0;
    }

    if (buffer[(y*rows + c_aux)*4+1] == 1 && buffer[(y*rows + c_aux)*4 + 3] == 1){
       array[tId*4+2] = 1;
    }else if (buffer[(y*rows + c_aux)*4] == 1 && buffer[(y*rows + c_aux)*4 + 2] == 1){
      array[tId*4+2] = 0;
    }else if (buffer[(y*rows + c_aux)*4+ 2] == 1){
        array[tId*4+2] = 1;
    }else if (buffer[(y*rows + c_aux)*4+ 2] == 0){
      array[tId*4+2] = 0;
    }

    c_aux = (((y-1)%rows)+rows)%rows*cols;

    if (buffer[(c_aux + x)*4] == 1 && buffer[(c_aux + x)*4+2] == 1){
       array[tId*4+1] = 1;
    }else if (buffer[(c_aux + x)*4+1] == 1 && buffer[(c_aux + x)*4+3] == 1){
      array[tId*4+1] = 0;
    }else if (buffer[ (c_aux + x)*4 + 1 ] == 1){
      array[tId*4+1] = 1;
    }else if (buffer[ (c_aux + x)*4 + 1 ] == 0){
      array[tId*4+1] = 0;
    }

    c_aux = (((y+1)%rows)*cols);

    if (buffer[(c_aux + x)*4] == 1 && buffer[(c_aux + x)*4 + 2] == 1){
       array[tId*4+3] = 1;
    }else if (buffer[(c_aux + x)*4+1] == 1 && buffer[(c_aux + x)*4 + 3] == 1){
      array[tId*4+3] = 0;
    }else if (buffer[ (c_aux + x)*4 + 3 ] == 1){
      array[tId*4+3] = 1;
    }else if (buffer[ (c_aux + x)*4 +3] == 0){
      array[tId*4+3] = 0;
    }
  }
}

int main(int argc, char const *argv[])
{
  int rows, cols;
  int *array;
  int *d_array;
  int *d_buffer;
  readInput_aos("../initial.txt", &array, &rows, &cols);

  int n = (int)(rows*cols);
  int block_size = 256;
  int grid_size = (int) ceil((float) n/ block_size);

  hipMalloc(&d_array ,4*rows * cols * sizeof(int));
  hipMalloc(&d_buffer,4*rows*cols*sizeof(int));
  hipMemcpy(d_array, array,4* rows * cols * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_buffer, array,4* rows * cols * sizeof(int), hipMemcpyHostToDevice);
  for(int k = 0; k < 1000; k++){
    step_periodic<<<grid_size, block_size>>>(d_array, d_buffer, rows, cols);
    hipMemcpy(d_buffer,d_array,4*rows*cols * sizeof(int), hipMemcpyDeviceToDevice);
  }
  hipMemcpy(array, d_array, 4*rows * cols * sizeof(int), hipMemcpyDeviceToHost);
  hipFree(d_array);
  hipFree(d_buffer);

  return(0);
}
