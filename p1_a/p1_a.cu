
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

void printMatrix(const int *A, int rows, int cols) {
    for(int i = 0; i < rows*cols*4; i++){
        printf("%d ", A[i]);   
        printf(" ");
        if ((i+1)%4 == 0){
          printf("|");
        }
    }
    printf("\n");
};

void readInput_aos(const char *filename, int **Aos, int *rows, int *cols) {
  FILE *file;
  file = fopen(filename, "r");
  fscanf(file, "%d %d", rows, cols);
  int * A_F1 = (int *) malloc(*rows * (*cols)* (4) * sizeof(int));
  for(int j = 0; j <  4; j++) {
    int counter = 0;
    for(int i = 0; i < *cols*(*rows); i++){
      fscanf(file, "%d ", &A_F1[counter +j]);
      counter = counter + 4;
      }
  }
  fclose(file);
  *Aos = A_F1;
}

__global__ void step_periodic_Aos(int * array,int rows, int cols){
  extern __shared__ int buffer[];
  int tId = threadIdx.x + blockIdx.x * blockDim.x;
  if(threadIdx.x < 256){
    for(int i = threadIdx.x; i < rows*cols; i+=256 ){
      if (array[i*4+0] == 1 && array[i*4+2] == 1){
        if(array[i*4+1] == 0 && array[i*4+3] == 0){
          buffer[i*4+0] = 0;
          buffer[i*4+2] = 0;
          buffer[i*4+1] = 1;
          buffer[i*4+3] = 1;
        }
      }else if(array[i*4+1] == 1 && array[i*4+3] == 1){
        if(array[i*4+0] == 0 && array[i*4+2] == 0){
          buffer[i*4+1] = 0;
          buffer[i*4+3] = 0;
          buffer[i*4+0] = 1;
          buffer[i*4+2] = 1;
          }
        }else{
          buffer[i*4+1] = array[i*4+1];
          buffer[i*4+3] = array[i*4+3];
          buffer[i*4+0] = array[i*4+0];
          buffer[i*4+2] = array[i*4+2];
      }
    }
  }
  __syncthreads();

  //if(tId == 1){
  //  for(int i = 0; i < rows*cols*4;i++){
  //    printf("%d ", buffer[i]);
  //    printf(" ");
  //  if ((i+1)%4 == 0){
  //        printf("|");
  //    }
 // }
  //printf("\n");
//}
  if (tId < rows*cols){
    int x = tId%(cols);
    int y = (int) tId/rows;

    int c_aux = x -1;
    if (c_aux < 0){
      c_aux = cols -1;
    }
    if (buffer[(y*rows + c_aux)*4] == 1){
      array[tId*4] = 1;
    }else if (buffer[(y*rows + c_aux)*4] == 0){
      array[tId*4] = 0;
    }

    c_aux = x + 1;
    if (c_aux == cols){
      c_aux = 0;
    }
    if (buffer[(y*rows + c_aux)*4+2] == 1){
      array[tId*4+2] = 1;
    }else if (buffer[(y*rows + c_aux)*4+2] == 0){
      array[tId*4+2] = 0;
    }

    //top
    c_aux = y - 1;
    if (c_aux <0){
      c_aux = rows-1;
    }
    if (buffer[(c_aux*rows + x)*4 + 1] == 1){
      array[tId*4+1] = 1;
    }else if (buffer[(c_aux*rows + x)*4+1] == 0){
      array[tId*4+1] = 0;
    }

    //bottom
    c_aux = y + 1;
    if (c_aux == rows){
      c_aux = 0;
    }
    if (buffer[(c_aux*rows + x)*4 + 3] == 1){
      array[tId*4+3] = 1;
    }else if(buffer[(c_aux*rows + x)*4+3] == 0){
      array[tId*4+3] = 0;
    }
  }
}

int main(int argc, char const *argv[])
{
  int rows, cols;
  int *Aos;
  int *d_Aos;

  readInput_aos("../initial.txt", &Aos, &rows, &cols);

  int n = (int)(rows*cols);
  int block_size = 256;
  int grid_size = (int) ceil((float)n / block_size);

  hipMalloc(&d_Aos, 4 * rows * cols * sizeof(int));
  hipMemcpy(d_Aos, Aos, 4 * rows * cols * sizeof(int), hipMemcpyHostToDevice);

  for(int k = 0; k < 1000; k++){
    step_periodic_Aos<<<grid_size, block_size,rows*cols*4>>>(d_Aos, rows, cols);
  }

  hipMemcpy(Aos, d_Aos, 4 * rows * cols * sizeof(int), hipMemcpyDeviceToHost);
  hipFree(d_Aos);

return 0;

}