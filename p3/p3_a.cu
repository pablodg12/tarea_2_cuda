
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

void readInput(const char *filename, int **Aos, int *rows, int *cols) {
  FILE *file;
  file = fopen(filename, "r");
  
  fscanf(file, "%d %d", rows, cols);
  int * A_F1 = (int *) malloc(*rows * (*cols)* (4) * sizeof(int));
  int * A_F2 = (int *) malloc(*rows * (*cols) * sizeof(int));
  for(int j = 0; j <  4; j++) {
    int counter = 0;
    for(int i = 0; i < *cols*(*rows); i++){
      fscanf(file, "%d ", &A_F1[counter +j]);
      counter = counter + 4;
      }
  }
  int counter = 0;
  for(int j = 0; j < *cols*(*rows);j++){
    A_F2[j] = A_F1[counter]*1 + A_F1[counter+1]*2 + A_F1[counter+2]*2*2 + A_F1[counter+3]*2*2*2;
    counter = counter +4;
  }
  *Aos = A_F2;
}

void printMatrix(int *A, int rows, int cols) {
    for(int i = 0; i < rows*cols; i++){
        printf("%i ", A[i]);
    }
    printf("\n");
};

__global__ void step_periodic(int * array,int *buffer,int rows, int cols){
  int tId = threadIdx.x + blockIdx.x * blockDim.x;
  if (tId < rows*cols){
    int c_aux;
    int reject = 1;
    int x = tId%(cols);
    int y = (int) tId/rows;
    int total = 0;


    c_aux = (x-1 < 0 ? cols-1 : x-1);
    reject = (x-1 < 0 ? 0:1);

    total = (reject ==1? (buffer[(y*cols + c_aux)]==1?(buffer[(y*cols + c_aux)] == 3?(buffer[(y*cols + c_aux)] == 10? (buffer[(y*cols + c_aux)] == 9?(buffer[(y*cols + c_aux)]==7?(buffer[(y*cols + c_aux)]==11?(buffer[(y*cols + c_aux)]==13?(buffer[(y*cols + c_aux)] == 15? total+1:0):0):0):0):0):0):0):0):0);
    total = (c_aux==0? (buffer[(y*cols + c_aux)]==4?(buffer[(y*cols + c_aux)] == 10?(buffer[(y*cols + c_aux)] == 6? (buffer[(y*cols + c_aux)] == 12?(buffer[(y*cols + c_aux)]==7?(buffer[(y*cols + c_aux)]==13?(buffer[(y*cols + c_aux)]==14?(buffer[(y*cols + c_aux)] == 15? total+1:0):0):0):0):0):0):0):0):0);

    c_aux = (x+1 == cols ? 0: x+1);
    reject = (x+1 == cols ? 0:1);

    total = (reject ==1? (buffer[(y*cols + c_aux)]==4?(buffer[(y*cols + c_aux)] == 10?(buffer[(y*cols + c_aux)] == 6? (buffer[(y*cols + c_aux)] == 12?(buffer[(y*cols + c_aux)]==7?(buffer[(y*cols + c_aux)]==13?(buffer[(y*cols + c_aux)]==14?(buffer[(y*cols + c_aux)] == 15? total+4:0):0):0):0):0):0):0):0):0);
    total = (c_aux==0? (buffer[(y*cols + c_aux)]==1?(buffer[(y*cols + c_aux)] == 3?(buffer[(y*cols + c_aux)] == 10? (buffer[(y*cols + c_aux)] == 9?(buffer[(y*cols + c_aux)]==7?(buffer[(y*cols + c_aux)]==11?(buffer[(y*cols + c_aux)]==13?(buffer[(y*cols + c_aux)] == 15? total+4:0):0):0):0):0):0):0):0):0);

    c_aux = (((y+1)%rows)*cols);
    reject = (y+1 == cols ? 0:1);

    total = (reject ==1? (buffer[(c_aux + x)]==2?(buffer[(c_aux + x)] == 3?(buffer[(c_aux + x)] == 6? (buffer[(c_aux + x)] == 5?(buffer[(c_aux + x)]==7?(buffer[(c_aux + x)]==11?(buffer[(c_aux + x)]==14?(buffer[(c_aux + x)] == 15? total+2:0):0):0):0):0):0):0):0):0);
    total = (c_aux==0? (buffer[(c_aux + x)]==8?(buffer[(c_aux + x)] == 12?(buffer[(c_aux + x)] == 5? (buffer[(c_aux + x)] == 9?(buffer[(c_aux + x)]==14?(buffer[(c_aux + x)]==13?(buffer[(c_aux + x)]==11?(buffer[(c_aux + x)] == 15? total+2:0):0):0):0):0):0):0):0):0);

    c_aux = (((y-1)%rows)+rows)%rows*cols;
    reject = (y-1 < 0 ? 0:1);


    total = (c_aux==0? (buffer[(c_aux + x)]==2?(buffer[(c_aux + x)] == 3?(buffer[(c_aux + x)] == 6? (buffer[(c_aux + x)] == 5?(buffer[(c_aux + x)]==7?(buffer[(c_aux + x)]==11?(buffer[(c_aux + x)]==14?(buffer[(c_aux + x)] == 15? total+2:0):0):0):0):0):0):0):0):0);
    total = (reject ==1? (buffer[(c_aux + x)]==8?(buffer[(c_aux + x)] == 12?(buffer[(c_aux + x)] == 5? (buffer[(c_aux + x)] == 9?(buffer[(c_aux + x)]==14?(buffer[(c_aux + x)]==13?(buffer[(c_aux + x)]==11?(buffer[(c_aux + x)] == 15? total+2:0):0):0):0):0):0):0):0):0);

    array[tId] = total;
  }
}
int main(int argc, char const *argv[])
{
  int rows, cols;
  int *array;
  int *d_array;
  int *d_buffer;
  readInput("../initial.txt", &array, &rows, &cols);

  int n = (int)(rows*cols);
  int block_size = 256;
  int grid_size = (int) ceil((float) n/ block_size);

  hipMalloc(&d_array ,rows * cols * sizeof(int));
  hipMalloc(&d_buffer,rows*cols*sizeof(int));
  hipMemcpy(d_array, array, rows * cols * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_buffer, array, rows * cols * sizeof(int), hipMemcpyHostToDevice);
  for(int k = 0; k < 1000; k++){
    step_periodic<<<grid_size, block_size>>>(d_array, d_buffer, rows, cols);
    hipMemcpy(d_buffer,d_array,rows*cols * sizeof(int), hipMemcpyDeviceToDevice);
  }
  hipMemcpy(array, d_array, rows * cols * sizeof(int), hipMemcpyDeviceToHost);
  hipFree(d_array);
  hipFree(d_buffer);

  return(0);
}
