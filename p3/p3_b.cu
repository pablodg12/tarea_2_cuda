
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

void readInput(const char *filename, int **Aos, int *rows, int *cols) {
  FILE *file;
  file = fopen(filename, "r");
  
  fscanf(file, "%d %d", rows, cols);
  int * A_F1 = (int *) malloc(*rows * (*cols)* (4) * sizeof(int));
  int * A_F2 = (int *) malloc(*rows * (*cols) * sizeof(int));
  for(int j = 0; j <  4; j++) {
    int counter = 0;
    for(int i = 0; i < *cols*(*rows); i++){
      fscanf(file, "%d ", &A_F1[counter +j]);
      counter = counter + 4;
      }
  }
  int counter = 0;
  for(int j = 0; j < *cols*(*rows);j++){
    A_F2[j] = A_F1[counter]*1 + A_F1[counter+1]*2 + A_F1[counter+2]*2*2 + A_F1[counter+3]*2*2*2;
    counter = counter +4;
  }
  *Aos = A_F2;
}

void printMatrix(int *A, int rows, int cols) {
    for(int i = 0; i < rows*cols; i++){
        printf("%i ", A[i]);
    }
    printf("\n");
};

__global__ void step_periodic(int * array,int rows, int cols){
  extern __shared__ int buffer[];
  int tId = threadIdx.x + blockIdx.x * blockDim.x;
  if(threadIdx.x < 256){
    for(int i = threadIdx.x; i < rows*cols; i+=256 ){
      int x = i%(cols);
      int y = (int) i/rows;
      buffer[i] = (array[i] == 10)*(x!=0)*(x!=cols-1)*(y != 0)*(y!= rows-1)*5 + (array[i] == 5)*(x!=0)*(x!=cols-1)*(y != 0)*(y!= rows-1)*10 + array[i]*(array[i] != 10)*(array[i] != 5);
    }
  }
   __syncthreads();

  if (tId < rows*cols){
    int reject;
    int x = tId%(cols);
    int y = (int) tId/rows;
    int total = 0;
	  int c_aux;
    c_aux = (x-1)*(x-1 > 0) + (cols-1)*(x-1 < 0);
    reject = (1)*(x-1 > 0) + (0)*(x-1 < 0);
    if (reject ==1 &&buffer[(y*rows + c_aux)] == 1 || buffer[(y*rows + c_aux)] == 3 || buffer[(y*rows + c_aux)] == 5 || 
        buffer[(y*rows + c_aux)] == 9 || buffer[(y*rows + c_aux)] == 7 || buffer[(y*rows + c_aux)] == 11 || 
        buffer[(y*rows + c_aux)] == 13 || buffer[(y*rows + c_aux)] == 15){
      total = total + 1;
    }else if(c_aux == 0){
    		if (buffer[(y*rows + c_aux)] == 4 || buffer[(y*rows + c_aux)] == 5 || buffer[(y*rows + c_aux)] == 6 || 
        buffer[(y*rows + c_aux)] == 12 || buffer[(y*rows + c_aux)] == 7 || buffer[(y*rows + c_aux)] == 13 || 
        buffer[(y*rows + c_aux)] == 14 || buffer[(y*rows + c_aux)] == 15){
    		total = total + 1;	
    		}
    	}else {
      total = total + 0;
    }

    c_aux = (0)*(x+1 == cols) + (x+1)*(x+1 != cols);
    reject = (0)*(x+1 == cols) + (1)*(x+1 != cols);

    if (reject ==1 &&buffer[(y*rows + c_aux)] == 4 || buffer[(y*rows + c_aux)] == 5 || buffer[(y*rows + c_aux)] == 6 || 
        buffer[(y*rows + c_aux)] == 12 || buffer[(y*rows + c_aux)] == 7 || buffer[(y*rows + c_aux)] == 13 || 
        buffer[(y*rows + c_aux)] == 14 || buffer[(y*rows + c_aux)] == 15){
      total = total + 4;
    }else if(c_aux == cols-1){
    		if (buffer[(y*rows + c_aux)] == 1 || buffer[(y*rows + c_aux)] == 3 || buffer[(y*rows + c_aux)] == 5 || 
        buffer[(y*rows + c_aux)] == 9 || buffer[(y*rows + c_aux)] == 7 || buffer[(y*rows + c_aux)] == 11 || 
        buffer[(y*rows + c_aux)] == 13 || buffer[(y*rows + c_aux)] == 15){
    		total = total + 4;	
    		}
    	}else {
      total = total + 0;
    }


    c_aux = (0)*(y+1 == rows) + (y+1)*(y+1 != rows);
    reject = (0)*(y+1 == rows) + (1)*(y+1 != rows);

    if (reject ==1 &&buffer[(c_aux*rows + x)] == 2 || buffer[(c_aux*rows + x)] == 3 || buffer[(c_aux*rows + x)] == 6 || 
        buffer[(c_aux*rows + x)] == 10 || buffer[(c_aux*rows + x)] == 7 || buffer[(c_aux*rows + x)] == 11 || 
        buffer[(c_aux*rows + x)] == 14 || buffer[(c_aux*rows + x)] == 15){
      total = total + 2;
    }else if(c_aux == cols-1){
    		if (buffer[(c_aux*rows + x)] == 8 || buffer[(c_aux*rows + x)] == 12 || buffer[(c_aux*rows + x)] == 10 || 
        buffer[(c_aux*rows + x)] == 9 || buffer[(c_aux*rows + x)] == 14 || buffer[(c_aux*rows + x)] == 13 || 
        buffer[(c_aux*rows + x)] == 11 || buffer[(c_aux*rows + x)] == 15){
    		total = total + 2;	
    		}
    	}else {
      total = total + 0;
    }

    c_aux = (rows-1)*(y-1 <0) + (y-1)*(y-1 > 0);
    reject = (0)*(y-1 <0) + (1)*(y-1 > 0);

    if (reject ==1 && buffer[(c_aux*rows + x)] == 8 || buffer[(c_aux*rows + x)] == 12 || buffer[(c_aux*rows + x)] == 10 || 
        buffer[(c_aux*rows + x)] == 9 || buffer[(c_aux*rows + x)] == 14 || buffer[(c_aux*rows + x)] == 13 || 
        buffer[(c_aux*rows + x)] == 11 || buffer[(c_aux*rows + x)] == 15){
        total = total + 8;
    }else if(c_aux == 0){
    		if (buffer[(c_aux*rows + x)] == 2 || buffer[(c_aux*rows + x)] == 3 || buffer[(c_aux*rows + x)] == 6 || 
        buffer[(c_aux*rows + x)] == 10 || buffer[(c_aux*rows + x)] == 7 || buffer[(c_aux*rows + x)] == 11 || 
        buffer[(c_aux*rows + x)] == 14 || buffer[(c_aux*rows + x)] == 15 ){
    		total = total + 8;	
    		}
    	}else{
      total = total + 0;
    }

    array[tId] = total;    
  }
  /* if(tId == 1){
    for(int i = 0; i < rows*cols;i++){
      printf("%d ", array[i]);
      if((i+1)%3==0){
      	printf("\n");
      }
    }
    printf("\n");
  }
  */
}

int main(int argc, char const *argv[])
{
  int rows, cols;
  int *array;
  int *d_array;

  readInput("../initial.txt", &array, &rows, &cols);
  //printMatrix(array,rows,cols);

  int n = (int)(rows*cols);
  int block_size = 256;
  int grid_size = (int) ceil((float)n / block_size);

  hipMalloc(&d_array ,rows * cols * sizeof(int));
  hipMemcpy(d_array, array, rows * cols * sizeof(int), hipMemcpyHostToDevice);
  for(int k = 0; k < 1000; k++){
    step_periodic<<<grid_size, block_size,rows*cols>>>(d_array, rows, cols);
  }
  hipMemcpy(array, d_array, rows * cols * sizeof(int), hipMemcpyDeviceToHost);
  hipFree(d_array);

  return(0);
}
