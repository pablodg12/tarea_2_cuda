
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

void readInput(const char *filename, int **Aos, int *rows, int *cols) {
  FILE *file;
  file = fopen(filename, "r");
  
  fscanf(file, "%d %d", rows, cols);
  int * A_F1 = (int *) malloc(*rows * (*cols)* (4) * sizeof(int));
  int * A_F2 = (int *) malloc(*rows * (*cols) * sizeof(int));
  for(int j = 0; j <  4; j++) {
    int counter = 0;
    for(int i = 0; i < *cols*(*rows); i++){
      fscanf(file, "%d ", &A_F1[counter +j]);
      counter = counter + 4;
      }
  }
  int counter = 0;
  for(int j = 0; j < *cols*(*rows);j++){
    A_F2[j] = A_F1[counter]*1 + A_F1[counter+1]*2 + A_F1[counter+2]*2*2 + A_F1[counter+3]*2*2*2;
    counter = counter +4;
  }
  *Aos = A_F2;
}

void printMatrix(int *A, int rows, int cols) {
    for(int i = 0; i < rows*cols; i++){
        printf("%i ", A[i]);
    }
    printf("\n");
};

__global__ void step_periodic(int * array,int *buffer,int rows, int cols){
  int tId = threadIdx.x + blockIdx.x * blockDim.x;
  if (tId < rows*cols){
    int reject = 1;
    int x = tId%(cols);
    int y = (int) tId/rows;
    int total = 0;


    int c_aux = x -1;
    if (c_aux < 0){
      c_aux = cols-1;
      reject = 0;
    }
    if (reject ==1 && buffer[(y*rows + c_aux)] == 1 || buffer[(y*rows + c_aux)] == 3 || buffer[(y*rows + c_aux)] == 10 || 
        buffer[(y*rows + c_aux)] == 9 || buffer[(y*rows + c_aux)] == 7 || buffer[(y*rows + c_aux)] == 11 || 
        buffer[(y*rows + c_aux)] == 13 || buffer[(y*rows + c_aux)] == 15){
      total = total + 1;
    }else if(c_aux == 0){
    		if (buffer[(y*rows + c_aux)] == 4 || buffer[(y*rows + c_aux)] == 10 || buffer[(y*rows + c_aux)] == 6 || 
        buffer[(y*rows + c_aux)] == 12 || buffer[(y*rows + c_aux)] == 7 || buffer[(y*rows + c_aux)] == 13 || 
        buffer[(y*rows + c_aux)] == 14 || buffer[(y*rows + c_aux)] == 15){
    		total = total + 1;	
    		}
    	}else {
      total = total + 0;
    }


    reject = 1;
    c_aux = x + 1;
    if (c_aux == cols){
      c_aux = 0;
      reject = 0;
    }
    if (reject ==1 && buffer[(y*rows + c_aux)] == 4 || buffer[(y*rows + c_aux)] == 10 || buffer[(y*rows + c_aux)] == 6 || 
        buffer[(y*rows + c_aux)] == 12 || buffer[(y*rows + c_aux)] == 7 || buffer[(y*rows + c_aux)] == 13 || 
        buffer[(y*rows + c_aux)] == 14 || buffer[(y*rows + c_aux)] == 15){
      total = total + 4;
    }else if(c_aux == cols-1){
    		if (buffer[(y*rows + c_aux)] == 1 || buffer[(y*rows + c_aux)] == 3 || buffer[(y*rows + c_aux)] == 10 || 
        buffer[(y*rows + c_aux)] == 9 || buffer[(y*rows + c_aux)] == 7 || buffer[(y*rows + c_aux)] == 11 || 
        buffer[(y*rows + c_aux)] == 13 || buffer[(y*rows + c_aux)] == 15){
    		total = total + 4;	
    		}
    	}else {
      total = total + 0;
    }


    reject = 1;
    c_aux = y + 1;
    if (c_aux == rows){
      c_aux = 0;
      reject = 0;
    }
    int g = (((y+1)%rows)*cols); 
    if (reject ==1 && buffer[(g + x)] == 2 || buffer[(g + x)] == 3 || buffer[(g + x)] == 6 || 
        buffer[(g + x)] == 5 || buffer[(g + x)] == 7 || buffer[(g + x)] == 11 || 
        buffer[(g + x)] == 14 || buffer[(g + x)] == 15){
      total = total + 2;
    }else if(c_aux == rows-1){
    		if (buffer[(g + x)] == 8 || buffer[(g + x)] == 12 || buffer[(g + x)] == 5 || 
        buffer[(g + x)] == 9 || buffer[(g + x)] == 14 || buffer[(g + x)] == 13 || 
        buffer[(g + x)] == 11 || buffer[(g + x)] == 15){
    		total = total + 2;	
    		}
    	}else {
      total = total + 0;
    }
    reject = 1;
    c_aux = y - 1;
    if (c_aux <0){
      c_aux = ((rows-1)%rows)*cols;
      reject = 0;
    }
    g = (((y-1)%rows)+rows)%rows;
    c_aux = g*cols;
    if (reject ==1 && buffer[(c_aux + x)] == 8 || buffer[(c_aux + x)] == 12 || buffer[(c_aux + x)] == 5 || 
        buffer[(c_aux + x)] == 9 || buffer[(c_aux + x)] == 14 || buffer[(c_aux + x)] == 13 || 
        buffer[(c_aux + x)] == 11 || buffer[(c_aux + x)] == 15){
        total = total + 8;
    }else if(c_aux == 0){
    		if (buffer[(c_aux + x)] == 2 || buffer[(c_aux + x)] == 3 || buffer[(c_aux + x)] == 6 || 
        buffer[(c_aux + x)] == 5 || buffer[(c_aux + x)] == 7 || buffer[(c_aux + x)] == 11 || 
        buffer[(c_aux + x)] == 14 || buffer[(c_aux + x)] == 15 ){
    		total = total + 8;	
    		}
    	}else{
      total = total + 0;
    }
    array[tId] = total;
  }
}

int main(int argc, char const *argv[])
{
  int rows, cols;
  int *array;
  int *d_array;
  int *d_buffer;
  readInput("../initial.txt", &array, &rows, &cols);
  //printMatrix(array,rows,cols);

  int n = (int)(rows*cols);
  int block_size = 256;
  int grid_size = (int) ceil((float) n/ block_size);

  hipMalloc(&d_array ,rows * cols * sizeof(int));
  hipMalloc(&d_buffer,rows*cols*sizeof(int));
  hipMemcpy(d_array, array, rows * cols * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_buffer, array, rows * cols * sizeof(int), hipMemcpyHostToDevice);
  for(int k = 0; k < 1000; k++){
    step_periodic<<<grid_size, block_size>>>(d_array, d_buffer, rows, cols);
    hipMemcpy(d_buffer,d_array,rows*cols * sizeof(int), hipMemcpyDeviceToDevice);
  }
  hipMemcpy(array, d_array, rows * cols * sizeof(int), hipMemcpyDeviceToHost);
  hipFree(d_array);
  hipFree(d_buffer);

  return(0);
}